extern "C"{
    #include "structures.h"
    #include "desicion_maker.h"
    #include "kernels.h"
}

#define COVERING_THREAD_PER_BLOCK 1024

int sum_array(int *a_in, int size)
{
    int sum = a_in[0];
    for(int i = 1 ; i < size ; i++)
        sum += a_in[i];
    return sum;
}

void run_bfs(struct graph * g_h)
{
    /* initial data on host */
    struct queue * workset = construct_queue(g_h->size);
    g_h->node_level_vector[0] = 0;
    queue_push(workset, 0);

    /* set and define desicion variables */
    int level = 0, block_count, thread_per_block, workset_size = workset->size;
    double avrage_outdeg = get_average_out_deg(g_h);
    int algo = decide(avrage_outdeg, workset_size, &block_count, &thread_per_block);
    int covering_block_count = (g_h->size - 1)/COVERING_THREAD_PER_BLOCK + 1;
    int update_size = covering_block_count * COVERING_THREAD_PER_BLOCK;
    int * add_result_h;
    add_result_h = (int *)malloc(sizeof(int)*covering_block_count);
    
    /* initial on and transform data to device */
    /*    initial workset queue on device      */
    struct queue * workset_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&workset_d, sizeof(struct queue *)));
    CUDA_CHECK_RETURN(hipMalloc( (void **)&workset_d->items, g_h->size));
    /*    transform workset queue to device    */
    CUDA_CHECK_RETURN(hipMemcpy(workset_d->items, workset->items, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&workset_d->size, &workset->size, sizeof(int), hipMemcpyHostToDevice));
    /*    initial graph on device    */
    struct graph * g_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d, sizeof(struct graph *)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d->size, sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d->node_vector, sizeof(int)*(g_h->size+1)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d->edge_vector, sizeof(int)*(g_h->node_vector[g->size])));
    //TODO: MACRO for initialing level or distance array for GPU
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d->node_level_vector, sizeof(int)*g_h->size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d->node_distance_vector, sizeof(int)*g_h->size));
    /*    transform graph to device  */
    CUDA_CHECK_RETURN(hipMemcpy(g_d->node_vector, g_h->node_vector, sizeof(int)*(g_h->size+1), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(g_d->edge_vector, g_h->edge_vector, sizeof(int)*(g_h->node_vector[g->size]), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(g_d->size, g_h->size, sizeof(int), hipMemcpyHostToDevice));
    /*    initial arrays on device    */
    char * update_d, * bitmap_d;
    int * add_result_d;
    int one = 1;
    CUDA_CHECK_RETURN(hipMalloc((void **)&update_d, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&bitmap_d, sizeof(char)*g_h->size));
    //TODO: initial zero on GPU update_d and bitmap_d (copy a zero-array to GPU or run a kernel for it)
    CUDA_CHECK_RETURN(hipMalloc((void **)&add_result_d, sizeof(int)*covering_block_count));
    /*    transform bitmap to device    */
    CUDA_CHECK_RETURN(hipMemcpy(bitmap_d, &one, sizeof(int), hipMemcpyHostToDevice));

    while (workset_size != 0)
    {
        if (algo == B_QU)
        {
            workset_update_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, workset_d);
            one_bfs_B_QU<<<block_count, thread_per_block>>>(g_d, workset_d, update_d, level++);
        } else if (algo == B_BM) 
        {
            workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);
            one_bfs_B_BM<<<block_count, thread_per_block>>>(g_d, bitmap_d, update_d, level++);
        } else if (algo == T_QU)
        {
            workset_update_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, workset_d);
            one_bfs_T_QU<<<block_count, thread_per_block>>>(g_d, workset_d, update_d, level++);
        } else if (algo == T_BM)
        {
            workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);
            one_bfs_T_BM<<<block_count, thread_per_block>>>(g_d, bitmap_d, update_d, level++);
        }
        /* calculate workset size and decide the next move */
        add_kernel<<<covering_block_count, COVERING_THREAD_PER_BLOCK, sizeof(int)*COVERING_THREAD_PER_BLOCK>>>(update_d, add_result_d);

        CUDA_CHECK_RETURN(hipDeviceSynchronize()); //wait for GPU
        CUDA_CHECK_RETURN(hipGetLastError());
        
        CUDA_CHECK_RETURN(hipMemcpy(add_result_h, add_result_d, sizeof(int)*covering_block_count, hipMemcpyDeviceToHost));
        workset_size = sum_array(add_result_h, covering_block_count);

        algo = decide(avrage_outdeg, workset_size, &block_count, &thread_per_block);
    }

    /*    free memory CPU and GPU    */
    //TODO
}

int main()
{
    return 0;
}