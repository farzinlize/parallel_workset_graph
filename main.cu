#include "kernels.cuh"
#include <stdio.h>
#include <limits.h>
#include "structures.h"
#include "sequential.h"
#include "report.h"
#include <limits.h>

extern "C"{
    #include "desicion_maker.h"
    #include "fuzzy_timing.h"
}

#define COVERING_THREAD_PER_BLOCK 1024
#define DATASET_COUNT 1

extern FILE * fileout;

const char * dataset_files[DATASET_COUNT][2] = {{"dataset/twitter-all.nodes", "dataset/twitter-all.edges"}};

int sum_array(int *a_in, int size)
{
    int sum = a_in[0];
    for(int i = 1 ; i < size ; i++)
        sum += a_in[i];
    return sum;
}

void T_BM_bfs(graph g_h, int source)
{
    /* necessary but not useful variables */
    #ifndef DP
    int one = 1, zero = 0;
    #endif

    /* set and define desicion variables */
    #ifndef DP
    int level = 0, workset_size = 1;
    #endif
    int covering_block_count = (g_h.size - 1)/COVERING_THREAD_PER_BLOCK + 1;
    int update_size = covering_block_count * COVERING_THREAD_PER_BLOCK;

    /* set reduction add kernel variables */
    int add_half_full_flag = covering_block_count%2;
    int add_block_count, add_block_size;
    if(add_half_full_flag){
        add_block_size = COVERING_THREAD_PER_BLOCK/2;
        add_block_count = covering_block_count;
    }else{
        add_block_size = COVERING_THREAD_PER_BLOCK;
        add_block_count = covering_block_count/2;
    }
    #ifndef DP
    int shared_size = add_block_size * sizeof(int);
    int * add_result_h = (int *)malloc(sizeof(int)*add_block_count);
    #endif

    /* initial graph on device based on BFS */
    graph g_d = consturct_graph_device(g_h);
    CUDA_CHECK_RETURN(hipMalloc((void **)&(g_d.node_level_vector), sizeof(int)*g_h.size));
    CUDA_CHECK_RETURN(hipMemset(g_d.node_level_vector, 20000, sizeof(int)*g_h.size));
    
    #ifdef DEBUG
    fprintf(fileout, "[DEBUG][INT_MAX] size of int in cpu: %d\n", sizeof(int));
    fprintf(fileout, "[DEBUG][INT_MAX] levels initialed with value of : 20000\n");
    fprintf(fileout, "[DEBUG][T_BM_BFS] graph successfully initialed on device\n");
    #endif

    /* initial arrays on device */
    char * update_d, * bitmap_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&update_d, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&bitmap_d, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMemset(update_d, 0, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMemset(bitmap_d, 0, sizeof(char)*update_size));

    int * add_result_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&(add_result_d), sizeof(int) * add_block_count));

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG][T_BM_BFS] arrays successfully initialed on device\n");
    #endif

    #ifndef DP
    /* bfs first move in butmap and level vector */
    //TODO: use hipMemset instead of copy or a better way
    CUDA_CHECK_RETURN(hipMemcpy(&bitmap_d[source], &one, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&g_d.node_level_vector[source], &zero, sizeof(int), hipMemcpyHostToDevice));
    #endif

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG][T_BM_BFS] first manual bfs move successfully done\n");
    #endif

    #ifndef DP
    while(workset_size != 0){
        one_bfs_T_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(g_d, bitmap_d, update_d, ++level);
        workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);

        #ifdef DEBUG
        fprintf(fileout, "[DEBUG][T_BM_BFS] bfs kernels of level:%d is launched\n", level);
        #endif

        if(add_half_full_flag){
            add_kernel_half<<<add_block_count, add_block_size, shared_size>>>(bitmap_d, add_result_d);
        }else{
            add_kernel_full<<<add_block_count, add_block_size, shared_size>>>(bitmap_d, add_result_d);
        }

        CUDA_CHECK_RETURN(hipDeviceSynchronize()); //wait for GPU
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(hipMemcpy(add_result_h, add_result_d, add_block_count*sizeof(int), hipMemcpyDeviceToHost));
        workset_size = sum_array(add_result_h, add_block_count);

        #ifdef DEBUG
        fprintf(fileout, "[DEBUG][T_BM_BFS] workset_size = %d\n", workset_size);
        #endif
    }
    #else
    argument argument_d;
    argument_d.covering_block_count = covering_block_count;
    argument_d.covering_block_size = COVERING_THREAD_PER_BLOCK;
    argument_d.add_half_full_flag = add_half_full_flag;
    argument_d.add_block_count = add_block_count;
    argument_d.add_block_size = add_block_size;
    argument_d.add_result = add_result_d;

    T_BM_bfs<<<1, 1>>>(g_d, source, bitmap_d, update_d, argument_d);

    CUDA_CHECK_RETURN(hipDeviceSynchronize()); //wait for GPU
    CUDA_CHECK_RETURN(hipGetLastError());
    #endif

    /* return level array of graph to host */
    CUDA_CHECK_RETURN(hipMemcpy(g_h.node_level_vector, g_d.node_level_vector, sizeof(int)*g_h.size, hipMemcpyDeviceToHost));

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG][T_BM_BFS] node level vector successfully returned to CPU\n");
    #endif

    /* free memory GPU */
    destroy_graph_device(g_d);
    hipFree(g_d.node_level_vector);
    hipFree(update_d);
    hipFree(bitmap_d);
    hipFree(add_result_d);

    /* free memory CPU */
    #ifndef DP
    free(add_result_h);
    #endif
}

void adaptive_bfs(graph g_h, int source)
{
    /* necessary but not useful variables */
    int one = 1, zero = 0;

    /* initial workset queue on device (instantly add first bfs move) */
    int workset_size_h = 1;
    queue workset_d = construct_queue_device_with_source(g_h.size, &source);

    /* set and define desicion variables */
    int level = 0;
    double avrage_outdeg = get_average_out_deg(g_h);
    int algo = decide(avrage_outdeg, workset_size_h);
    int next_sample = next_sample_distance();
    int covering_block_count = (g_h.size - 1)/COVERING_THREAD_PER_BLOCK + 1;
    int update_size = covering_block_count * COVERING_THREAD_PER_BLOCK;
    int * add_result_h = (int *)malloc(sizeof(int)*covering_block_count);

    /* initial graph on device based on BFS */
    graph g_d = consturct_graph_device(g_h);
    CUDA_CHECK_RETURN(hipMalloc((void **)&(g_d.node_level_vector), sizeof(int)*g_h.size));
    CUDA_CHECK_RETURN(hipMemset(g_d.node_level_vector, INT_MAX, sizeof(int)*g_h.size));
    
    /* initial arrays on device */
    char * update_d, * bitmap_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&update_d, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&bitmap_d, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMemset(update_d, 0, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMemset(bitmap_d, 0, sizeof(char)*update_size));

    int * add_result_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&add_result_d, sizeof(int)*covering_block_count));

    /* bfs first move (workset updated instantly after initialized) */
    CUDA_CHECK_RETURN(hipMemcpy(&bitmap_d[source], &one, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&g_d.node_level_vector[source], &zero, sizeof(int), hipMemcpyHostToDevice));

    while (workset_size_h != 0)
    {
        if (algo == B_QU)
        {
            while(next_sample--)
            {
                workset_update_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, workset_d);
                one_bfs_B_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(g_d, workset_d, update_d, level++);
            }
        } else if (algo == B_BM) 
        {
            while(next_sample--)
            {
                workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);
                one_bfs_B_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(g_d, bitmap_d, update_d, level++);
            }
        } else if (algo == T_QU)
        {
            while(next_sample--)
            {
                workset_update_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, workset_d);
                one_bfs_T_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(g_d, workset_d, update_d, level++);
            }
        } else if (algo == T_BM)
        {
            while(next_sample--)
            {
                workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);
                one_bfs_T_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(g_d, bitmap_d, update_d, level++);
            }
        }
        /* calculate workset size and decide the next move */
        add_kernel<<<covering_block_count, COVERING_THREAD_PER_BLOCK, sizeof(int)*COVERING_THREAD_PER_BLOCK>>>(update_d, add_result_d);

        CUDA_CHECK_RETURN(hipDeviceSynchronize()); //wait for GPU
        CUDA_CHECK_RETURN(hipGetLastError());
        
        CUDA_CHECK_RETURN(hipMemcpy(add_result_h, add_result_d, sizeof(int)*covering_block_count, hipMemcpyDeviceToHost));
        workset_size_h = sum_array(add_result_h, covering_block_count);

        algo = decide(avrage_outdeg, workset_size_h);
        next_sample = next_sample_distance();
    }

    /* return level array of graph to host */
    CUDA_CHECK_RETURN(hipMemcpy(g_h.node_level_vector, g_d.node_level_vector, sizeof(int)*g_h.size, hipMemcpyDeviceToHost));

    /* free memory GPU */
    destroy_queue_device(workset_d);
    destroy_graph_device(g_d);
    hipFree(g_d.node_level_vector);
    hipFree(update_d);
    hipFree(bitmap_d);
    hipFree(add_result_d);

    /* free memory CPU */
    free(add_result_h);
}

#ifndef TEST
int main(int argc, char * argv[])
{
    initial_fileout();
    fprintf(fileout, "[MAIN] app.cu main\tDataset index: %d\n", DATASET_INDEX);

    /* read data set */
    graph g_h = consturct_graph(dataset_files[DATASET_INDEX][0], dataset_files[DATASET_INDEX][1]);

    /* initial bfs arrays */
    g_h.node_level_vector = (int *)malloc(sizeof(int)*g_h.size);

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG][MAIN] running sequential bfs with graph size: %d\n", g_h.size);
    #endif

    /* sequentinal run */
    set_clock();
    sequential_run_bfs_QU(&g_h, 0);
    double elapced = get_elapsed_time();

    fprintf(fileout, "[MAIN] returning sequential bfs, time: %.2f\n", elapced);

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG] first 10 nodes level (sequentianl):\n");
    for(int i=0;i<10;i++){
        fprintf(fileout, "node %d | level %d\n", i, g_h.node_level_vector[i]);
    }
    #endif

    /* Save sequential result for future use */
    int * sequential_result = (int *)malloc(sizeof(int)*g_h.size);
    memcpy(sequential_result, g_h.node_level_vector, sizeof(int)*g_h.size);

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG] first 10 sequential result:\n");
    for(int i=0;i<10;i++){
        fprintf(fileout, "node %d | level %d\n", i, g_h.node_level_vector[i]);
    }
    #endif

    /* parallel run (T_BM) */
    set_clock();
    T_BM_bfs(g_h, 0);
    elapced = get_elapsed_time();

    fprintf(fileout, "[MAIN] returning parallel (T_BM) bfs, time: %.2f\n", elapced);

    #ifdef DEBUG
    fprintf(fileout, "[DEBUG] first 10 nodes level (parallel):\n");
    for(int i=0;i<10;i++){
        fprintf(fileout, "node %d | level %d\n", i, g_h.node_level_vector[i]);
    }
    #endif

    /* make compare files */
    make_compare_file("out/compare_seq_TBM.out", "sequentinal", sequential_result, "T_BM", g_h.node_level_vector, g_h.size);

    /* free allocated memory in main function */
    free(g_h.node_level_vector);
    destroy_graph(g_h);

    return 0;
}
#else

int main(int argc, char * argv[])
{
    int a[5] = {5, 3, 2, 1, 9};
    int b[5];

    memcpy(b, a, sizeof(a));

    for(int i=0;i<5;i++){
        printf("b[%d]=%d\t", i, b[i]);
    }

    return 0;
}
#endif
