extern "C"{
    #include "structures.h"
    #include "desicion_maker.h"
    #include "kernels.h"
    #include <limits.h>
}

#define COVERING_THREAD_PER_BLOCK 1024

int sum_array(int *a_in, int size)
{
    int sum = a_in[0];
    for(int i = 1 ; i < size ; i++)
        sum += a_in[i];
    return sum;
}

void run_bfs(struct graph * g_h, int source)
{
    /* necessary but not useful variables */
    int one = 1, zero = 0;

    /* initial workset queue on device */
    struct queue * workset_d = construct_queue_device(g_h->size);
    int workset_size = 0
    queue_push_device(workset_d, source, &workset_size);

    /* set and define desicion variables */
    int level = 0, block_count, thread_per_block;
    double avrage_outdeg = get_average_out_deg(g_h);
    int algo = decide(avrage_outdeg, workset_size, &block_count, &thread_per_block);
    int next_sample = next_sample_distance();
    int covering_block_count = (g_h->size - 1)/COVERING_THREAD_PER_BLOCK + 1;
    int update_size = covering_block_count * COVERING_THREAD_PER_BLOCK;
    int * add_result_h = (int *)malloc(sizeof(int)*covering_block_count);

    /* initial graph on device based on BFS */
    struct graph * g_d = consturct_graph_device(g_h);
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_d->node_level_vector, sizeof(int)*g_h->size));
    inital_int_array<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(g_d->node_level_vector, INT_MAX, g_h->size);
    
    /* initial arrays on device */
    char * update_d, * bitmap_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&update_d, sizeof(char)*update_size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&bitmap_d, sizeof(char)*g_h->size));
    inital_char_array<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, 0);
    inital_char_array<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(bitmap_d, 0);

    int * add_result_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&add_result_d, sizeof(int)*covering_block_count));

    CUDA_CHECK_RETURN(hipDeviceSynchronize()); //wait for inital kernels
    CUDA_CHECK_RETURN(hipGetLastError());

    /* bfs first move (workset updated instantly after initialized) */
    CUDA_CHECK_RETURN(hipMemcpy(&bitmap_d[source], &one, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&g_d->node_level_vector[source], &zero, sizeof(int), hipMemcpyHostToDevice));

    while (workset_size != 0)
    {
        if (algo == B_QU)
        {
            while(next_sample--)
            {
                workset_update_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, workset_d);
                one_bfs_B_QU<<<block_count, thread_per_block>>>(g_d, workset_d, update_d, level++);
            }
        } else if (algo == B_BM) 
        {
            while(next_sample--)
            {
                workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);
                one_bfs_B_BM<<<block_count, thread_per_block>>>(g_d, bitmap_d, update_d, level++);
            }
        } else if (algo == T_QU)
        {
            while(next_sample--)
            {
                workset_update_QU<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, workset_d);
                one_bfs_T_QU<<<block_count, thread_per_block>>>(g_d, workset_d, update_d, level++);
            }
        } else if (algo == T_BM)
        {
            while(next_sample--)
            {
                workset_update_BM<<<covering_block_count, COVERING_THREAD_PER_BLOCK>>>(update_d, bitmap_d);
                one_bfs_T_BM<<<block_count, thread_per_block>>>(g_d, bitmap_d, update_d, level++);
            }
        }
        /* calculate workset size and decide the next move */
        add_kernel<<<covering_block_count, COVERING_THREAD_PER_BLOCK, sizeof(int)*COVERING_THREAD_PER_BLOCK>>>(update_d, add_result_d);

        CUDA_CHECK_RETURN(hipDeviceSynchronize()); //wait for GPU
        CUDA_CHECK_RETURN(hipGetLastError());
        
        CUDA_CHECK_RETURN(hipMemcpy(add_result_h, add_result_d, sizeof(int)*covering_block_count, hipMemcpyDeviceToHost));
        workset_size = sum_array(add_result_h, covering_block_count);

        algo = decide(avrage_outdeg, workset_size, &block_count, &thread_per_block);
        next_sample = next_sample_distance();
    }

    /* return level array of graph to host */
    CUDA_CHECK_RETURN(hipMemcpy(g_h->node_level_vector, g_d->node_level_vector, sizeof(int)*g->size, hipMemcpyDeviceToHost));

    /* free memory GPU */
    destroy_queue_device(workset_d);
    destroy_graph_device(g_d);
    hipFree(update_d);
    hipFree(bitmap_d);
    hipFree(add_result_d);

    /* free memory CPU */
    free(add_result_h);
}

int main()
{
    return 0;
}